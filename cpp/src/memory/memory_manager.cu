/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "cucim/memory/memory_manager.h"

#include <fmt/format.h>
#include <hip/hip_runtime.h>

#include <memory_resource>

#define CUDA_TRY(stmt)                                                                                                 \
    {                                                                                                                  \
        cuda_status = stmt;                                                                                            \
        if (hipSuccess != cuda_status)                                                                                \
        {                                                                                                              \
            fmt::print(stderr, "[Error] CUDA Runtime call {} in line {} of file {} failed with '{}' ({}).\n", #stmt,   \
                       __LINE__, __FILE__, hipGetErrorString(cuda_status), cuda_status);                              \
        }                                                                                                              \
    }

CUCIM_API void* cucim_malloc(size_t size)
{
    return malloc(size);
}

CUCIM_API void cucim_free(void* ptr)
{
    free(ptr);
}

namespace cucim::memory
{

void get_pointer_attributes(PointerAttributes& attr, const void* ptr)
{
    hipError_t cuda_status;

    hipPointerAttribute_t attributes;
    CUDA_TRY(hipPointerGetAttributes(&attributes, ptr));
    if (cuda_status)
    {
        return;
    }

    hipMemoryType& memory_type = attributes.type;
    switch (memory_type)
    {
    case cudaMemoryTypeUnregistered:
        attr.device = cucim::io::Device(cucim::io::DeviceType::kCPU, -1);
        attr.ptr = const_cast<void*>(ptr);
        break;
    case hipMemoryTypeHost:
        attr.device = cucim::io::Device(cucim::io::DeviceType::kPinned, attributes.device);
        attr.ptr = attributes.hostPointer;
        break;
    case hipMemoryTypeDevice:
    case hipMemoryTypeManaged:
        attr.device = cucim::io::Device(cucim::io::DeviceType::kCUDA, attributes.device);
        attr.ptr = attributes.devicePointer;
        break;
    }
}

} // namespace cucim::memory